#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <cmath>
#include <fstream>
#include <sstream>

#define NUM_BLOCKS 1
#define THREADS_PER_BLOCK 32

using namespace std;

//-----Unified Memory Class Constructor; all shared memory classes inherit------

class Managed {
public:
    void *operator new(size_t len) {
        void *ptr;
        hipMallocManaged(&ptr, len);
        hipDeviceSynchronize();
        return ptr;
    }

    void operator delete(void *ptr) {
        hipDeviceSynchronize();
        hipFree(ptr);
    }
};

//-----------------------Training Class to load training data-------------------

class TrainingData : public Managed
{
public:
    TrainingData(const string filename);
    ~TrainingData();
    bool isEof(void) { return m_trainingDataFile.eof(); }
    void getTopology(vector<unsigned> &topology);

    // Returns the number of input values read from the file:
    unsigned getNextInputs(vector<double> &inputVals);
    unsigned getTargetOutputs(vector<double> &targetOutputVals);

private:
    ifstream m_trainingDataFile;
};

void TrainingData::getTopology(vector<unsigned> &topology)
{
    string line;
    string label;

    getline(m_trainingDataFile, line);
    stringstream ss(line);
    ss >> label;
    if (this->isEof() || label.compare("topology:") != 0) {
        abort();
    }

    while (!ss.eof()) {
        unsigned n;
        ss >> n;
        topology.push_back(n);
    }

    return;
}

TrainingData::TrainingData(const string filename)
{
    m_trainingDataFile.open(filename.c_str());
}
TrainingData::~TrainingData(const string filename)
{
    m_trainingDataFile.close();
}

unsigned TrainingData::getNextInputs(vector<double> &inputVals)
{
    inputVals.clear();

    string line;
    getline(m_trainingDataFile, line);
    stringstream ss(line);

    string label;
    ss>> label;
    if (label.compare("in:") == 0) {
        double oneValue;
        while (ss >> oneValue) {
            inputVals.push_back(oneValue);
        }
    }

    return inputVals.size();
}

unsigned TrainingData::getTargetOutputs(vector<double> &targetOutputVals)
{
    targetOutputVals.clear();

    string line;
    getline(m_trainingDataFile, line);
    stringstream ss(line);

    string label;
    ss>> label;
    if (label.compare("out:") == 0) {
        double oneValue;
        while (ss >> oneValue) {
            targetOutputVals.push_back(oneValue);
        }
    }

    return targetOutputVals.size();
}

//----------------------------------Neural Net----------------------------------

struct Connection{
    double weight;
    double deltaWeight;
};

class Neuron;

typedef vector<Neuron> Layer;

// Neuron class
class Neuron : public Managed {
public:
    Neuron(unsigned numOutputs, unsigned myIndex);
    void setOutputVal(double val){m_outputVal = val;}
    double getOutputVal(void) const{return m_outputVal;}
    void feedforward(const Layer &prevLayer);
    void calculateOutputGradients(double targetVal);
    void calculateHiddenGradients(const Layer &nextlayer);
    void updateInputWeights(Layer &prevlayer);
private:
    static double eta; // overall learning rate [0.0-1.0]
    static double alpha; //multiplier of last weight change (momentum) [0.0 - n or 1]
    static double transferFunction(double x);
    static double transferFunctionDerivative(double x);
    static double randomWeight(void) {return rand()/double(RAND_MAX);}
    double sumDOW(const Layer &nextlayer) const;
    double m_outputVal;
    vector<Connection> m_outputWeights;
    unsigned m_myIndex;
    double m_gradient;
};

double Neuron::eta = 0.15;
double Neuron::alpha = 0.5;

//This function updates the weights of each neuron in the layer
//prevLayer is the layer to be updated
__global__ void Neuron::d_updateInputWeights(Layer &prevLayer){
    for(unsigned n = blockIdx.x*blockDim.x +  threadIdx.x;
        n < prevLayer.size();
        n += blockDim.x * gridDim.x)
    {
        Neuron &neuron = prevlayer[n];
        double oldDeltaWeight = neuron.m_outputWeights[m_myIndex].deltaWeight;
        //individual weight, magnified by gradient and train rate, then add momentum
        double newDeltaWeight = eta * neuron.getOutputVal() * m_gradient + alpha * oldDeltaWeight;

        neuron.m_outputWeights[m_myIndex].deltaWeight = newDeltaWeight;
        neuron.m_outputWeights[m_myIndex].weight += newDeltaWeight;
    }
}
__device__ void Neuron::updateInputWeights(Layer &prevlayer){
    //the weights to be updated are in the connection container in the neurons of the preceding layer
    Neuron::d_updateInputWeights<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (prevlayer);
    hipDeviceSynchronize();
}

__global__ void Neuron::d_sumDOW(double *sum, Layer nextLayer){
    for(unsigned n = blockIdx.x*blockDim.x +  threadIdx.x;
        n < nextLayer.size()-1;
        n += blockDim.x * gridDim.x)
    {
        *sum += m_outputWeights[n].weight * nextLayer[n].m_gradient;
    }
}
__device__ double Neuron::sumDOW(const Layer &nextlayer)const{
    double sum = 0.0;
    //sum our contributions of the errors at the nodes we feed
    Neuron::d_sumDOW<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (&sum, nextlayer);
    hipDeviceSynchronize();
    return sum;
}

__global__ void Neuron::calculateHiddenGradients(const Layer &nextlayer){
    double dow = sumDOW(nextlayer);
    m_gradient = dow * Neuron::transferFunctionDerivative(m_outputVal);
}
__device__ void Neuron::calculateOutputGradients(double targetVal){
    double delta = targetVal - m_outputVal;
    m_gradient = delta * Neuron::transferFunctionDerivative(m_outputVal);
}

__device__ double Neuron::transferFunctionDerivative(double x){
    return 1.0 - x * x;
}
__device__ double Neuron::transferFunction(double x){
    ///tanh - output range [-1.0, 1.0]
    return tanh(x);
}

__global__ void Neuron::d_feedForward(double *sum, Layer &prevLayer){
    for(unsigned n = blockIdx.x*blockDim.x +  threadIdx.x;
        n < prevLayer.size();
        n += blockDim.x * gridDim.x)
    {
        *sum += prevLayer[n].getOutputVal() * prevLayer[n].m_outputWeights[m_myIndex].weight;
    }
}
__device__ void Neuron::feedforward(const Layer & prevLayer){
    double sum = 0.0;

    //sum the previous layer outputs (which are our inputs)
    // include bias node from previous layer
    Neuron::d_feedForward<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (&sum, prevLayer);
    hipDeviceSynchronize();
    m_outputVal = Neuron::transferFunction(sum);
}

__device__ Neuron::Neuron(unsigned numOutputs, unsigned myIndex){
    for (unsigned c = 0; c < numOutputs; ++c){
        m_outputWeights.push_back(Connection());
        m_outputWeights.back().weight = randomWeight();
    }
    m_myIndex = myIndex;
}

class Net : public Managed {
public:
    Net(const vector<unsigned> &topology);
    void feedforward(const vector<double> &inputVals);
    void backprop(const vector<double> &targetVals);
    void getResults(vector<double> &resultVals) const;
    double getRecentAverageError(void) const { return m_recentAverageError; }


private:
    vector<Layer> m_layers; //m_layers[layerNum][neuronNum]
    double m_error;
    double m_recentAverageError;
    static double m_recentAverageSmoothing;
};

double Net::m_recentAverageSmoothing = 100.0; //Number of training samples to average over

__device__ void Net::getResults(vector<double> &resultVals) const{

    resultVals.clear();

    for(unsigned n = 0; n < m_layers.back().size()-1; ++n){

        resultVals.push_back(m_layers.back()[n].getOutputVal());
    }

}

__global__ void Net::hidden_backprop(Layer &hiddenLayer, Layer &nextLayer){
    for(unsigned n = blockIdx.x*blockDim.x +  threadIdx.x;
        n < hiddenLayer.size();
        n += blockDim.x * gridDim.x)
    {
        hiddenlayer[n].calculateHiddenGradients(nextlayer);
    }
}
__global__ void Net::update_weights(Layer layer, Layer prevLayer){
    for(unsigned n = blockIdx.x*blockDim.x +  threadIdx.x;
        n < layer.size()-1;
        n += blockDim.x * gridDim.x)
    {
        layer[n].updateInputWeights(prevLayer);
    }
}
__device__ void Net::backprop(const vector<double> &targetVals){
    // calculate overall net error (RMS of outputs neuron errors)
    Layer &outputLayer = m_layers.back();
    m_error = 0.0;
    for(unsigned n = 0; n < outputLayer.size()-1; ++n){
        double delta = targetVals[n] - outputLayer[n].getOutputVal();
        m_error += delta*delta;
    }
    m_error /= outputLayer.size()-1; //get average error squared
    m_error = sqrt(m_error); //RMS

    //Implement a recent average measurement

    m_recentAverageError = (m_recentAverageError * m_recentAverageSmoothing + m_error)
                            / (m_recentAverageSmoothing + 1.0);

    // Calculate output layer gradients
    for(unsigned n =0; n < outputLayer.size()-1; ++n){
        outputLayer[n].calculateOutputGradients(targetVals[n]);
    }

    // calculate gradients on hidden layers
    for(unsigned layerNum = m_layers.size()-2; layerNum > 0; --layerNum){
        Layer &hiddenlayer = m_layers[layerNum];
        Layer &nextlayer = m_layers[layerNum+1];

        Net::hidden_backprop<<<1, 32>>> (hiddenlayer, nextlayer);
        hipDeviceSynchronize();

        //For all layers from outputs to first hidden layer, update connection weights
        for(unsigned layerNum = m_layers.size()-1;layerNum > 0; --layerNum){
            Layer &layer = m_layers[layerNum];
            Layer &prevlayer = m_layers[layerNum-1];

            Net::update_weights<<<NUM_BLOCKS,THREADS_PER_BLOCK>>> (layer, prevLayer);
            hipDeviceSynchronize();
        }
    }
}

__global__ void Net::d_feedForward(Layer() &prevLayer, unsigned layerNum){
    for(unsigned n = blockIdx.x*blockDim.x +  threadIdx.x;
        n < m_layers[layerNum].size() - 1;
        n += blockDim.x * gridDim.x)
    {
        m_layers[layerNum][n].feedforward(prevLayer);
    }
}
__device__ void Net::feedforward(const vector<double> &inputVals){

    assert(inputVals.size() == m_layers[0].size() - 1);

    //assign the input values to the input neurons
    for(unsigned i = 0; i < inputVals.size(); ++i){
        m_layers[0][i].setOutputVal(inputVals[i]);
    }

    //forward prop
    for(unsigned layerNum = 1; layerNum < m_layers.size(); ++layerNum){
        Layer &prevLayer = m_layers[layerNum-1];
        Net::d_feedForward<<<1, 32>>> (prevLayer, layerNum);
        hipDeviceSynchronize();
    }
}

__device__ Net::Net(const vector<unsigned> &topology){
    unsigned numLayers = topology.size();
    for(unsigned layerNum = 0; layerNum < numLayers; ++layerNum){
        m_layers.push_back(Layer());
        unsigned numOutputs = layerNum == topology.size() - 1 ? 0 : topology[layerNum+1];

        //We have made a new layer, now fill it with ith neurons, and add
        //a bias neuron to the layer:
        for(unsigned neuronNum = 0; neuronNum <= topology[layerNum]; ++neuronNum){
            m_layers.back().push_back(Neuron(numOutputs, neuronNum));
        }
        //force bias to be 1.0 output
        m_layers.back().back().setOutputVal(1.0);
    }
}


__device__ void showVectorVals(string label, vector<double> &v)
{
    cout << label << " ";
    for (unsigned i = 0; i < v.size(); ++i) {
        cout << v[i] << " ";
    }
    cout << endl;
}



int main(){
    TrainingData trainData("trainingdata.txt");

    vector<unsigned> topology;
    trainData.getTopology(topology);

    Net myNet(topology);

    vector<double> inputVals, targetVals, resultVals;
    int trainingPass = 0;

    while (!trainData.isEof()) {
        ++trainingPass;
        cout << endl << "Pass " << trainingPass;

        // Get new input data and feed it forward:
        if (trainData.getNextInputs(inputVals) != topology[0]) {
            break;
        }
        showVectorVals(": Inputs:", inputVals);
        myNet.feedforward(inputVals);

        // Collect the net's actual output results:
        myNet.getResults(resultVals);
        showVectorVals("Outputs:", resultVals);

        // Train the net what the outputs should have been:
        trainData.getTargetOutputs(targetVals);
        showVectorVals("Targets:", targetVals);
        assert(targetVals.size() == topology.back());
        myNet.backprop(targetVals);

        // Report how well the training is working, average over recent samples:
        cout << "Net recent average error: "
                << myNet.getRecentAverageError() << endl;
    }
    cout << endl << "Done" << endl;
}
