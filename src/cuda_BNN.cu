#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cassert>
#include <cmath>
#include <fstream>
#include <sstream>
#include "training_reader.cuh"


#define NUM_BLOCKS 1
#define THREADS_PER_BLOCK 4
#define INPUT_SIZE 2
#define HIDDEN_SIZE 4
#define OUTPUT_SIZE 1
#define NUM_HIDDEN_LAYERS 1
#define MOMENTUM 0.5
#define LR 0.15

using namespace std;

//-----Unified Memory Class Constructor; all shared memory classes inherit------

class Managed {
    // Overwrites constructors so that all classes that inherit from
    //  Managed are created in unified memory, so can be used by both
    //  CPU and GPU
public:
    void *operator new(size_t len) {
        void *ptr;
        hipMallocManaged(&ptr, len);
        hipDeviceSynchronize();
        return ptr;
    }

    void operator delete(void *ptr) {
        hipDeviceSynchronize();
        hipFree(ptr);
    }
};



//----------------------------------Neural Declarations-------------------------
class Connection;
class Neuron;
class Layer;
class Network;

//--------------------------------Global Declarations---------------------------

__host__ void showVectorVals(string label, double *v, int length);
__global__ void neuron_global_feed_forward(Neuron *n, double *sum, Layer *prev_layer);
__global__ void neuron_global_sum_DOW(Neuron *n, double *sum, Layer *next_layer);
__global__ void neuron_global_update_input_weights(Neuron *n, Layer *prev_layer);
__global__ void net_global_feed_forward(Layer *layer, Layer *prev_layer);
__global__ void net_global_update_weights(Layer *layer, Layer *prev_layer);
__global__ void net_global_backprop(Layer *hidden_layer, Layer *next_layer);


//-------------------------------Net Class Initializations----------------------

class Connection: public Managed
{
    // weighted connection between two neurons on different layers.
    //  delta_weight (how much it should change) is calculated during
    //  backpropagation, based on the output error.
public:
    double weight;
    double delta_weight;
};


class Neuron: public Managed
{
    // Perceptron node that outputs the sum of all inputs times the weight
    //  of the connection between it and the neuron before, then passes
    //  this value through the connections to all neurons in the next layer.
public:
    __host__ Neuron();
    __host__ Neuron(int num_neurons, int num_connections);
    __host__ void set_output(double val){output = val;}
    __host__ __device__ double get_output(void) {return output;}
    __host__ __device__ void feed_forward(Layer *prev_layer);
    __host__ __device__ void calculate_output_gradient(double target_val);
    __host__ __device__ void calculate_hidden_gradients(Layer *next_layer);
    __host__ __device__ void update_input_weights(Layer *prev_layer);
    double output;
    Connection** output_weights;
    unsigned my_index;
    double gradient;
    double* DOW_sum;
    double* FF_sum;

private:

    __host__ __device__ static double transfer_function(double x);
    __host__ __device__ static double transfer_function_derivative(double x);
    static double init_weight(void) {return rand()/double(RAND_MAX);} // randomly assigns an initial weight for each connection
    __host__ __device__ double sum_DOW(Layer *next_layer);


};



class Layer: public Managed
{
    // Container for all the neurons in a layer. Acts as an array.
public:
    __host__ Layer();
    __host__ Layer(int num_neurons, int num_connections);
    Neuron** layer;
    int length;
};


class Network: public Managed
{
    // Container for all layers and wrapper for calls to individual neurons.
public:
    __host__ Network();
    __host__ void feed_forward(double *input_vals, int input_vals_length);
    __host__ void back_prop(double * target_vals, int target_length);

    __host__ void get_results(double *result_vals, int result_length);
    __host__ double get_RAE() const { return RAE; }

private:
    Layer **layers;
    double error;
    double RAE;
    static double RAS;

};
double Network::RAS = 100.0; //Number of training samples to average over


//------------------------------Global Functions--------------------------------

__host__
void showVectorVals(string label, double *v, int length)
{
    // Print out results nicely
    cout << label << " ";
    for (unsigned i = 0; i < length; ++i) {
        cout << v[i] << " ";
    }
    cout << endl;
}

__global__
void neuron_global_feed_forward(Neuron *neuron, double *sum, Layer *prev_layer)
{
    // Stands in as a neuron's sum. Gets all the outputs from the previous
    //  layer, multiplies them by the weights of the connections, and sums
    //  the results. The sum is assigned to the output of the given neuron.

    // In theory, this should be parallelizable for all neurons within the same
    //  layer, as neurons on the same layer do not effect each other. This did
    //  not seem to be the case.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int n = index; n < prev_layer->length; n+=stride) {
        *sum = *sum + prev_layer->layer[n]->get_output() *
                prev_layer->layer[n]->output_weights[neuron->my_index]->weight;
    }

}

__global__
void neuron_global_sum_DOW(Neuron *neuron, double *sum, Layer *next_layer)
{
    // Sums the Derivative of Weights, which will be used to calculate the
    //  gradient and adjust the weights of the connections for the next pass.

    // Should be parallelizable within a layer.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int n = index; n < next_layer->length - 1; n+=stride) {
        *sum = *sum + neuron->output_weights[n]->weight * next_layer->layer[n]->gradient;
    }

}
__global__
void neuron_global_update_input_weights(Neuron *neuron, Layer *prev_layer)
{
    // Based on the previous delta_weight and the gradient, updates the input
    //  weight to each neuron in order to minimize error from output.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int n = index; n < prev_layer->length; n+=stride) {
        Neuron* prev_neuron = prev_layer->layer[n];
        double old_delta_weight = prev_neuron->output_weights[neuron->my_index]->delta_weight;

        double new_delta_weight =
                // Individual input, magnified by the gradient and train rate:
                LR
                * prev_neuron->get_output()
                * neuron->gradient
                // Also add momentum = a fraction of the previous delta weight;
                + MOMENTUM
                * old_delta_weight;

        // Adjust the connections
        prev_neuron->output_weights[neuron->my_index]->delta_weight = new_delta_weight;
        prev_neuron->output_weights[neuron->my_index]->weight += new_delta_weight;

    }

}

__global__
void net_global_feed_forward(Layer *layer, Layer *prev_layer)
{
    // Wrapper around each step of the feed forward process. Given two sequential
    //  layers, iterates through the neurons and performs their feed_forward method.
    //  Each layer must run sequentially, and thus is not parallelizable.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i=index; i < layer->length-1;i+=stride){
        // Call to the neuron device function feed_forward.
        //  Ideally, all neuron feed_forwards are calculated simultaneously.
        layer->layer[i]->feed_forward(prev_layer);
    }

}

__global__
void net_global_update_weights(Layer *layer, Layer *prev_layer)
{
    // Wrapper around the update weight process. Given two sequential layers,
    //  iterates through the neurons and calculates the gradients.
    //  Each layer must run sequentially, and thus is not parallelizable.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=index; i < layer->length-1;i+=stride){
        // Call to the neuron device function update_input_weights.
        //  Ideally, all neuron calculate_hidden_gradients are calculated simultaneously
        layer->layer[i]->update_input_weights(prev_layer);
    }

}

__global__
void net_global_backprop(Layer *hidden_layer, Layer *next_layer)
{
    // Wrapper around the backpropagation. Given two sequential layers,
    //  iterates through the neurons and calculates the gradients.
    //  Each layer must run sequentially, and thus is not parallelizable.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int n = index; n < hidden_layer->length; n+=stride) {
        // Call to the neuron device function calculate_hidden_gradients.
        //  Ideally, all neuron calculate_hidden_gradients are calculated simultaneously
        hidden_layer->layer[n]->calculate_hidden_gradients(next_layer);
    }

}


//--------------------------Class Functions-------------------------------------
__host__
__device__
void Neuron::update_input_weights(Layer *prev_layer)
{
    // wrapper that calls update_input_weights, parallelizing them.
    neuron_global_update_input_weights<<<NUM_BLOCKS, THREADS_PER_BLOCK>>> (this, prev_layer);
    hipDeviceSynchronize();
}
__host__
__device__
double Neuron::sum_DOW(Layer *next_layer)
{
    // wrapper that calculates the derivative of weights based on the error
    *DOW_sum = 0.0;
    // Sum our contributions of the errors at the nodes we feed.

    neuron_global_sum_DOW<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(this, DOW_sum, next_layer);
    hipDeviceSynchronize();
    return *DOW_sum;
}
__host__
__device__
void Neuron::calculate_hidden_gradients(Layer *next_layer)
{
    // Uses the derivative of weights to calculate the gradient, which is used to update the weights.
    double dow = sum_DOW(next_layer);
    gradient = dow * Neuron::transfer_function_derivative(output);
}

__host__
__device__
void Neuron::calculate_output_gradient(double target_val)
{
    // Calculates error, then uses it to determine output gradients.
    double delta = target_val - output;
    gradient = delta * Neuron::transfer_function_derivative(output);
}

__host__
__device__
double Neuron::transfer_function_derivative(double x)
{
    // Derivative transfer function to calculate derivative of weights
    return 1.0 - x * x;
}
__host__
__device__
double Neuron::transfer_function(double x)
{
    // Transfer function to determine the output value
    return tanh(x);
}
__host__
__device__
void Neuron::feed_forward(Layer *prev_layer)
{
    // feed_forward wrapper
    *FF_sum = 0.0;

    neuron_global_feed_forward<<<1, 1>>>(this, FF_sum, prev_layer);
    hipDeviceSynchronize();
    output = Neuron::transfer_function(*FF_sum);
}

__host__
Neuron::Neuron()
{
   my_index = 999;
}
__host__
 Neuron::Neuron(int num_connections, int index)
{
    // Initializes a neuron and makes sure that all pointers are in
    //  unified memory
    hipMallocManaged(&output_weights, sizeof(Connection *)*num_connections);
    for (unsigned i = 0; i < num_connections; ++i){
        Connection* c;
        hipMallocManaged(&c, sizeof(Connection));
        *c = Connection();
        c->weight = Neuron::init_weight();

        output_weights[i] = c;
    }
    hipMallocManaged(&DOW_sum, sizeof(double));
    hipMallocManaged(&FF_sum, sizeof(double));
    *DOW_sum = 0.0;
    *FF_sum = 0.0;
    my_index = index;
}


__host__
Layer::Layer()
{
    length = 0;
}
__host__
Layer::Layer(int num_neurons, int num_connections)
{
    // Creates a new layer and ensures that everything is in unified memory.
    hipMallocManaged(&layer, sizeof(Neuron *)*num_neurons);
    for(int i=0;i<=num_neurons;i++){
        Neuron *n;
        hipMallocManaged(&n, sizeof(Neuron)); // possibly redundant
        *n = Neuron(num_connections, i);
        n->set_output(1.0);
        layer[i] = n;
    }
    length = num_neurons+1;
}


__host__
void Network::get_results(double *result_vals, int result_length)
{
    // Assigns the results to result_vals
    for(unsigned n = 0; n < result_length; ++n){
        Layer* output_layer = layers[NUM_HIDDEN_LAYERS+1];
        result_vals[n] = (output_layer->layer[n]->get_output());
    }
}

__host__
void Network::back_prop(double * target_vals, int target_length)
{
    // Performs backpropagation, making sure everything runs sequentially as needed.
    //  Each time hipDeviceSynchronize is called, we are switching between
    //  the host and device, which is a costly operation.
    Layer* output_layer = layers[NUM_HIDDEN_LAYERS+1];
    error = 0.0;
    for(unsigned n = 0; n < output_layer->length-1; ++n){
        double delta = target_vals[n] - output_layer->layer[n]->get_output();
        error += delta*delta;
    }
    error /= (output_layer->length-1); //get average error squared
    error = sqrt(error); //RMS

    RAE = (RAE * RAS + error) / (RAS + 1.0);

    // Calculate output layer gradients
    for(unsigned n =0; n < output_layer->length-1; ++n){
        output_layer->layer[n]->calculate_output_gradient(target_vals[n]);
    }

    // calculate gradients on hidden layers
    for(unsigned layer_num = NUM_HIDDEN_LAYERS; layer_num > 0; --layer_num){
        Layer* hidden_layer = layers[layer_num];
        Layer* next_layer = layers[layer_num+1];

        net_global_backprop<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(hidden_layer, next_layer);
        hipDeviceSynchronize();
    }

    //For all layers from outputs to first hidden layer, update connection weights
    for(unsigned layer_num = NUM_HIDDEN_LAYERS+1;layer_num > 0; --layer_num){
        Layer* layer = layers[layer_num];
        Layer* prev_layer = layers[layer_num-1];

        net_global_update_weights<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(layer, prev_layer);
        hipDeviceSynchronize();
    }

}

__host__
void Network::feed_forward(double *input_vals, int input_vals_length)
{

    //assign the input values to the input neurons
    for(unsigned i = 0; i < input_vals_length; ++i){
        Layer* input_layer = layers[0];
        input_layer->layer[i]->set_output(input_vals[i]);
    }


    //forward prop
    for(unsigned num_layer = 1; num_layer < NUM_HIDDEN_LAYERS+2; ++num_layer){
        Layer* layer = layers[num_layer];
        Layer* prev_layer = layers[num_layer-1];
        net_global_feed_forward<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(layer, prev_layer);
        hipDeviceSynchronize();
    }

}


__host__
Network::Network()
{
    // ensures all aspects are in unified memory
    hipMallocManaged(&layers, sizeof(Layer *)*(NUM_HIDDEN_LAYERS+2));
    Layer * layer;
    hipMallocManaged(&layer, sizeof(Layer));
    *layer = Layer(INPUT_SIZE, HIDDEN_SIZE);
    layers[0] = layer;
    for (int i = 1; i<NUM_HIDDEN_LAYERS; i++) {
        Layer * layer;
        hipMallocManaged(&layer, sizeof(Layer));
        *layer = Layer(HIDDEN_SIZE, HIDDEN_SIZE);
        layers[i] = layer;
    }
    Layer * layer_2;
    hipMallocManaged(&layer_2, sizeof(Layer));
    *layer_2 = Layer(HIDDEN_SIZE, OUTPUT_SIZE);
    layers[NUM_HIDDEN_LAYERS] = layer_2;
    Layer * layer_3;
    hipMallocManaged(&layer_3, sizeof(Layer));
    *layer_3 = Layer(OUTPUT_SIZE, 0);
    layers[1 + NUM_HIDDEN_LAYERS] = layer_3;
}



int main(){
    // Read training data
    TrainingData trainData("final_training_data.txt");

    // initialize network on host, but in unified memory
    Network myNet = Network();

    double input_vals[INPUT_SIZE];
    double target_vals[OUTPUT_SIZE];
    double result_vals[OUTPUT_SIZE];
    int training_pass = 0;

    while (!trainData.isEof()) {
        ++training_pass;
        cout << endl << "Pass " << training_pass;

        // Get new input data and feed it forward:
        trainData.getNextInputs(input_vals);

        // Get new input data and feed it forward:
        showVectorVals("Inputs:", input_vals, INPUT_SIZE);
        myNet.feed_forward(input_vals, INPUT_SIZE);

        // Collect the net's actual output results:
        myNet.get_results(result_vals, OUTPUT_SIZE);
        showVectorVals("Outputs:", result_vals, OUTPUT_SIZE);

        // Train the net what the outputs should have been:
        trainData.getTargetOutputs(target_vals);
        showVectorVals("Targets:", target_vals, OUTPUT_SIZE);
        myNet.back_prop(target_vals, OUTPUT_SIZE);

        // Report how well the training is working, average over recent samples:
        cout << "Net recent average error: " << myNet.get_RAE() << endl;
    }
    cout << endl << "Done!" << endl;
    return 0;
}
